#include "hip/hip_runtime.h"
﻿/**
* ARQUITECTURA DE COMPUTADORES
* 2º Grado en Ingenieria Informatica
*
* Básico 5
*
* Alumno: Rodrigo Pascual Arnaiz 
* Fecha: 19/10/2020
*
*/
///////////////////////////////////////////////////////////////////////////
// includes
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <>
///////////////////////////////////////////////////////////////////////////
// defines
#define PI 3.141593f

///////////////////////////////////////////////////////////////////////////
// declaracion de funciones
// HOST: funcion llamada desde el host y ejecutada en el host

/**
* Funcion: propiedadesDispositivo
* Objetivo: Mustra las propiedades del dispositvo, esta funcion
* es ejecutada llamada y ejecutada desde el host
*
* Param: INT id_dispositivo -> ID del dispotivo
* Return: void
* propiedades del dispositivo CUDA
*/
__host__ void propiedadesDispositivo(int id_dispositivo)
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, id_dispositivo);
	// calculo del numero de cores (SP)
	int cuda_cores = 0;
	int multi_processor_count = deviceProp.multiProcessorCount;
	int major = deviceProp.major;
	int minor = deviceProp.minor;
	char* arquitectura = (char*)"";
	switch (major)
	{
	case 1:
		//TESLA
		cuda_cores = 8;
		arquitectura = (char*)"TESLA";
		break;
	case 2:
		//FERMI
		arquitectura = (char*)"FERMI";
		if (minor == 0)
			cuda_cores = 32;
		else
			cuda_cores = 48;
		break;
	case 3:
		//KEPLER
		arquitectura = (char*)"KEPLER";
		cuda_cores = 192;
		break;
	case 5:
		//MAXWELL
		arquitectura = (char*)"MAXWELL";
		cuda_cores = 128;
		break;
	case 6:
		//PASCAL
		arquitectura = (char*)"PASCAL";
		cuda_cores = 64;
		break;
	case 7:
		//VOLTA
		arquitectura = (char*)"VOLTA";
		cuda_cores = 64;
		break;
	case 8:
		//AMPERE
		arquitectura = (char*)"AMPERE";
		cuda_cores = 128;
		break;
	default:
		arquitectura = (char*)"DESCONOCIDA";
		//DESCONOCIDA
		cuda_cores = 0;
		printf("!!!!!dispositivo desconocido!!!!!\n");
	}
	// presentacion de propiedades
	printf("***************************************************\n");
	printf("DEVICE %d: %s\n", id_dispositivo, deviceProp.name);
	printf("***************************************************\n");
	printf("> Capacidad de Computo \t\t\t: %d.%d\n", major, minor);
	printf("> Arquitectura CUDA \t\t\t: %s \n", arquitectura);
	printf("> No. de MultiProcesadores \t\t: %d \n",
		multi_processor_count);
	printf("> No. de CUDA Cores (%dx%d) \t\t: %d \n", cuda_cores,
		multi_processor_count, cuda_cores*
		multi_processor_count);
	printf("> No. max. de Hilos (por bloque) \t: %d \n",
		deviceProp.maxThreadsPerBlock);
	printf("***************************************************\n");

}

///////////////////////////////////////////////////////////////////////////
// KERNEL
/**
* Funcion: pi
* Objetivo: Funcion que calcula el numero pi utilizando reduccion paralela
*
* Param: INT  terminos->Cantidad de terminos 
* Param : INT * resultado->Puntero resultado 
* Param : INT * temporal->Array temporal para poder acceder a datos de otros hilos
* Return : void
*/
__global__ void pi(int terminos, float* resultado, float* temporal)
{
	// indice local de cada hilo -> kernel con un solo bloque de N hilos
	int my_id = threadIdx.x;
	// rellenamos el vector de datos aplicando cada uno de los terminos de la sucesion 
	temporal[my_id] =  ( 1 / pow( (my_id + 1.0), 2 ) );
	

	// sincronizamos para evitar riesgos de tipo RAW
	__syncthreads();

	// ******************
	// REDUCCION PARALELA
	// ******************
		int salto = terminos / 2;
		// realizamos log2(N) iteraciones
		while (salto > 0)
		{
			// en cada paso solo trabajan la mitad de los hilos
			if (my_id < salto)
			{
				temporal[my_id] = temporal[my_id] + temporal[my_id + salto];
			}
			// sincronizamos los hilos evitar riesgos de tipo RAW
			__syncthreads();
			salto /= 2;
		}

	// Solo el hilo no.'0' escribe el resultado final
	if (my_id == 0)
	{
		*resultado = sqrt( (temporal[0] * 6) );
	}


}
///////////////////////////////////////////////////////////////////////////
// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{
	
	// Declaración de variables
	int deviceCount;
	bool is_numero_valido = false;
	int numero_terminos;
	float hst_init = 0;
	float hst_result, * dev_result, * dev_temp;

	// Buscando dispositivos
	hipGetDeviceCount(&deviceCount);

	
	// Mostrar propiedades por pantalla
	if (deviceCount == 0)
	{
		printf("!!!!!No se han encontrado dispositivos CUDA!!!!!\n");
		printf("<pulsa [INTRO] para finalizar>");
		getchar();
		return 1;
	} 
	else
	{
		printf("Se han encontrado <%d> dispositivos CUDA:\n", deviceCount);
		for (int id = 0; id < deviceCount; id++)
		{
			propiedadesDispositivo(id);
		}
	}

	
	// Pedir numero de terminos para sacar el numero pi
	do {
		printf("  Introduce el numero de terminos (potencia de 2):");
		is_numero_valido = scanf("%i", &numero_terminos);
		printf("\n");
		if (is_numero_valido ) 
		{
			is_numero_valido = numero_terminos % 2 == 0 && numero_terminos > 0;
			if( !is_numero_valido )
				printf("  ERROR -> Tiene que ser potencia de 2 y mayor que 0\n");
		}
		else 
		{
			printf("  ERROR -> Tiene que ser un numero\n");
		}
	} while (!is_numero_valido);

	// reserva de memoria en el device
	hipMalloc((void**)&dev_result, sizeof(float));
	hipMalloc((void**)&dev_temp, sizeof(float) * numero_terminos);

	// Copiar datos al dispositivo
	hipMemcpy(dev_result, &hst_init, sizeof(float), hipMemcpyHostToDevice);
	
	// Ejecutamos la funcion PI con la cantidad de hilos siendo el numero de terminos introducido por el usuario 
	pi <<<1, numero_terminos >> > (numero_terminos, dev_result, dev_temp);

	// Copiar datos del dispositivo al host
	hipMemcpy(&hst_result, dev_result, sizeof(float), hipMemcpyDeviceToHost);


	// Mostramos los datos 
	printf("> Valor de pi:\t\t%f%\n", PI);
	printf("> Valor calculado:\t%f%\n", hst_result);
	printf("> Error relativo:\t%f%%\n", (hst_result * 100 / PI) - 100);


	// Salida del programa
	time_t fecha;
	time(&fecha);
	printf("***************************************************\n");
	printf("Programa ejecutado el: %s\n", ctime(&fecha));
	printf("<pulsa [INTRO] para finalizar>");
	getchar();
	return 0;
}
///////////////////////////////////////////////////////////////////////////
